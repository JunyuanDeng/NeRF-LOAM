#include "hip/hip_runtime.h"
#include "mc_data.cuh"

#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>

__device__ static inline float2 query_sdf_raw(uint bx, uint by, uint bz, uint arx, uint ary, uint arz,
                                              const uint max_vec_num,
                                              const IndexerAccessor indexer,
                                              const CubeSDFAccessor cube_sdf,
                                              const CubeSDFAccessor cube_std,
                                              const BackwardMappingAccessor vec_batch_mapping)
{
    if (bx >= indexer.size(0) || by >= indexer.size(1) || bz >= indexer.size(2))
    {
        return make_float2(NAN, NAN);
    }
    //    printf("B-Getting: %d %d %d --> %d, %d, %d\n", bx, by, bz, indexer.size(0), indexer.size(1), indexer.size(2));
    long long vec_ind = indexer[bx][by][bz];
    if (vec_ind == -1 || vec_ind >= max_vec_num)
    {
        return make_float2(NAN, NAN);
    }
    int batch_ind = vec_batch_mapping[vec_ind];
    if (batch_ind == -1)
    {
        return make_float2(NAN, NAN);
    }
    //    printf("Getting: %d %d %d %d --> %d %d\n", batch_ind, arx, ary, arz, cube_sdf.size(0), cube_sdf.size(1));
    float sdf = cube_sdf[batch_ind][arx][ary][arz];
    float std = cube_std[batch_ind][arx][ary][arz];
    return make_float2(sdf, std);
}

// Use stddev to weight sdf value.
// #define STD_W_SDF

__device__ static inline float2 get_sdf(const uint3 bsize, const uint r, uint3 bpos, uint3 rpos, const uint max_vec_num,
                                        const IndexerAccessor indexer,
                                        const CubeSDFAccessor cube_sdf,
                                        const CubeSDFAccessor cube_std,
                                        const BackwardMappingAccessor vec_batch_mapping)
{
    if (bpos.x >= bsize.x)
    {
        bpos.x = bsize.x - 1;
        rpos.x = r - 1;
    }
    if (bpos.y >= bsize.y)
    {
        bpos.y = bsize.y - 1;
        rpos.y = r - 1;
    }
    if (bpos.z >= bsize.z)
    {
        bpos.z = bsize.z - 1;
        rpos.z = r - 1;
    }

    uint rbound = (r - 1) / 2;
    uint rstart = r / 2;
    float rmid = r / 2.0f;

    float w_xm, w_xp;
    int bxm, rxm, bxp, rxp;
    int zero_x;
    if (rpos.x <= rbound)
    {
        bxm = -1;
        rxm = r;
        bxp = 0;
        rxp = 0;
        w_xp = (float)rpos.x + rmid;
        w_xm = rmid - (float)rpos.x;
        zero_x = 1;
    }
    else
    {
        bxm = 0;
        rxm = 0;
        bxp = 1;
        rxp = -r;
        w_xp = (float)rpos.x - rmid;
        w_xm = rmid + r - (float)rpos.x;
        zero_x = 0;
    }
    w_xm /= r;
    w_xp /= r;

    float w_ym, w_yp;
    int bym, rym, byp, ryp;
    int zero_y;
    if (rpos.y <= rbound)
    {
        bym = -1;
        rym = r;
        byp = 0;
        ryp = 0;
        w_yp = (float)rpos.y + rmid;
        w_ym = rmid - (float)rpos.y;
        zero_y = 1;
    }
    else
    {
        bym = 0;
        rym = 0;
        byp = 1;
        ryp = -r;
        w_yp = (float)rpos.y - rmid;
        w_ym = rmid + r - (float)rpos.y;
        zero_y = 0;
    }
    w_ym /= r;
    w_yp /= r;

    float w_zm, w_zp;
    int bzm, rzm, bzp, rzp;
    int zero_z;
    if (rpos.z <= rbound)
    {
        bzm = -1;
        rzm = r;
        bzp = 0;
        rzp = 0;
        w_zp = (float)rpos.z + rmid;
        w_zm = rmid - (float)rpos.z;
        zero_z = 1;
    }
    else
    {
        bzm = 0;
        rzm = 0;
        bzp = 1;
        rzp = -r;
        w_zp = (float)rpos.z - rmid;
        w_zm = rmid + r - (float)rpos.z;
        zero_z = 0;
    }
    w_zm /= r;
    w_zp /= r;

    rpos.x += rstart;
    rpos.y += rstart;
    rpos.z += rstart;

    // printf("%u %u %u %d %d %d %d %d %d\n", rpos.x, rpos.y, rpos.z, rxm, rxp, rym, ryp, rzm, rzp);

    // Tri-linear interpolation of SDF values.
#ifndef STD_W_SDF
    float total_weight = 0.0;
#else
    float2 total_weight{0.0, 0.0};
#endif
    float2 total_sdf{0.0, 0.0};

    int zero_det = zero_x * 4 + zero_y * 2 + zero_z;

    float2 sdfmmm = query_sdf_raw(bpos.x + bxm, bpos.y + bym, bpos.z + bzm, rpos.x + rxm, rpos.y + rym, rpos.z + rzm,
                                  max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    float wmmm = w_xm * w_ym * w_zm;
#ifndef STD_W_SDF
    if (!isnan(sdfmmm.x))
    {
        total_sdf += sdfmmm * wmmm;
        total_weight += wmmm;
    }
#else
    if (!isnan(sdfmmm.x))
    {
        total_sdf.x += sdfmmm.x * wmmm * sdfmmm.y;
        total_weight.x += wmmm * sdfmmm.y;
        total_sdf.y += wmmm * sdfmmm.y;
        total_weight.y += wmmm;
    }
#endif
    else if (zero_det == 0)
    {
        return make_float2(NAN, NAN);
    }

    float2 sdfmmp = query_sdf_raw(bpos.x + bxm, bpos.y + bym, bpos.z + bzp, rpos.x + rxm, rpos.y + rym, rpos.z + rzp,
                                  max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    float wmmp = w_xm * w_ym * w_zp;
#ifndef STD_W_SDF
    if (!isnan(sdfmmp.x))
    {
        total_sdf += sdfmmp * wmmp;
        total_weight += wmmp;
    }
#else
    if (!isnan(sdfmmp.x))
    {
        total_sdf.x += sdfmmp.x * wmmp * sdfmmp.y;
        total_weight.x += wmmp * sdfmmp.y;
        total_sdf.y += wmmp * sdfmmp.y;
        total_weight.y += wmmp;
    }
#endif
    else if (zero_det == 1)
    {
        return make_float2(NAN, NAN);
    }

    float2 sdfmpm = query_sdf_raw(bpos.x + bxm, bpos.y + byp, bpos.z + bzm, rpos.x + rxm, rpos.y + ryp, rpos.z + rzm,
                                  max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    float wmpm = w_xm * w_yp * w_zm;
#ifndef STD_W_SDF
    if (!isnan(sdfmpm.x))
    {
        total_sdf += sdfmpm * wmpm;
        total_weight += wmpm;
    }
#else
    if (!isnan(sdfmpm.x))
    {
        total_sdf.x += sdfmpm.x * wmpm * sdfmpm.y;
        total_weight.x += wmpm * sdfmpm.y;
        total_sdf.y += wmpm * sdfmpm.y;
        total_weight.y += wmpm;
    }
#endif
    else if (zero_det == 2)
    {
        return make_float2(NAN, NAN);
    }

    float2 sdfmpp = query_sdf_raw(bpos.x + bxm, bpos.y + byp, bpos.z + bzp, rpos.x + rxm, rpos.y + ryp, rpos.z + rzp,
                                  max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    float wmpp = w_xm * w_yp * w_zp;
#ifndef STD_W_SDF
    if (!isnan(sdfmpp.x))
    {
        total_sdf += sdfmpp * wmpp;
        total_weight += wmpp;
    }
#else
    if (!isnan(sdfmpp.x))
    {
        total_sdf.x += sdfmpp.x * wmpp * sdfmpp.y;
        total_weight.x += wmpp * sdfmpp.y;
        total_sdf.y += wmpp * sdfmpp.y;
        total_weight.y += wmpp;
    }
#endif
    else if (zero_det == 3)
    {
        return make_float2(NAN, NAN);
    }

    float2 sdfpmm = query_sdf_raw(bpos.x + bxp, bpos.y + bym, bpos.z + bzm, rpos.x + rxp, rpos.y + rym, rpos.z + rzm,
                                  max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    float wpmm = w_xp * w_ym * w_zm;
#ifndef STD_W_SDF
    if (!isnan(sdfpmm.x))
    {
        total_sdf += sdfpmm * wpmm;
        total_weight += wpmm;
    }
#else
    if (!isnan(sdfpmm.x))
    {
        total_sdf.x += sdfpmm.x * wpmm * sdfpmm.y;
        total_weight.x += wpmm * sdfpmm.y;
        total_sdf.y += wpmm * sdfpmm.y;
        total_weight.y += wpmm;
    }
#endif
    else if (zero_det == 4)
    {
        return make_float2(NAN, NAN);
    }

    float2 sdfpmp = query_sdf_raw(bpos.x + bxp, bpos.y + bym, bpos.z + bzp, rpos.x + rxp, rpos.y + rym, rpos.z + rzp,
                                  max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    float wpmp = w_xp * w_ym * w_zp;
#ifndef STD_W_SDF
    if (!isnan(sdfpmp.x))
    {
        total_sdf += sdfpmp * wpmp;
        total_weight += wpmp;
    }
#else
    if (!isnan(sdfpmp.x))
    {
        total_sdf.x += sdfpmp.x * wpmp * sdfpmp.y;
        total_weight.x += wpmp * sdfpmp.y;
        total_sdf.y += wpmp * sdfpmp.y;
        total_weight.y += wpmp;
    }
#endif
    else if (zero_det == 5)
    {
        return make_float2(NAN, NAN);
    }

    float2 sdfppm = query_sdf_raw(bpos.x + bxp, bpos.y + byp, bpos.z + bzm, rpos.x + rxp, rpos.y + ryp, rpos.z + rzm,
                                  max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    float wppm = w_xp * w_yp * w_zm;
#ifndef STD_W_SDF
    if (!isnan(sdfppm.x))
    {
        total_sdf += sdfppm * wppm;
        total_weight += wppm;
    }
#else
    if (!isnan(sdfppm.x))
    {
        total_sdf.x += sdfppm.x * wppm * sdfppm.y;
        total_weight.x += wppm * sdfppm.y;
        total_sdf.y += wppm * sdfppm.y;
        total_weight.y += wppm;
    }
#endif
    else if (zero_det == 6)
    {
        return make_float2(NAN, NAN);
    }

    float2 sdfppp = query_sdf_raw(bpos.x + bxp, bpos.y + byp, bpos.z + bzp, rpos.x + rxp, rpos.y + ryp, rpos.z + rzp,
                                  max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    float wppp = w_xp * w_yp * w_zp;
#ifndef STD_W_SDF
    if (!isnan(sdfppp.x))
    {
        total_sdf += sdfppp * wppp;
        total_weight += wppp;
    }
#else
    if (!isnan(sdfppp.x))
    {
        total_sdf.x += sdfppp.x * wppp * sdfppp.y;
        total_weight.x += wppp * sdfppp.y;
        total_sdf.y += wppp * sdfppp.y;
        total_weight.y += wppp;
    }
#endif
    else if (zero_det == 7)
    {
        return make_float2(NAN, NAN);
    }

    // If NAN, will also be handled.
    return total_sdf / total_weight;
}

__device__ static inline float4 sdf_interp(const float3 p1, const float3 p2, const float stdp1, const float stdp2,
                                           float valp1, float valp2)
{
    if (fabs(0.0f - valp1) < 1.0e-5f)
        return make_float4(p1, stdp1);
    if (fabs(0.0f - valp2) < 1.0e-5f)
        return make_float4(p2, stdp2);
    if (fabs(valp1 - valp2) < 1.0e-5f)
        return make_float4(p1, stdp1);

    float w2 = (0.0f - valp1) / (valp2 - valp1);
    float w1 = 1 - w2;

    return make_float4(p1.x * w1 + p2.x * w2,
                       p1.y * w1 + p2.y * w2,
                       p1.z * w1 + p2.z * w2,
                       stdp1 * w1 + stdp2 * w2);
}

__global__ static void meshing_cube(const IndexerAccessor indexer,
                                    const ValidBlocksAccessor valid_blocks,
                                    const BackwardMappingAccessor vec_batch_mapping,
                                    const CubeSDFAccessor cube_sdf,
                                    const CubeSDFAccessor cube_std,
                                    TrianglesAccessor triangles,
                                    TriangleStdAccessor triangle_std,
                                    TriangleVecIdAccessor triangle_flatten_id,
                                    int *__restrict__ triangles_count,
                                    int max_triangles_count,
                                    const uint max_vec_num,
                                    int nx, int ny, int nz,
                                    float max_std)
{
    const uint r = cube_sdf.size(1) / 2;
    const uint r3 = r * r * r;
    const uint num_lif = valid_blocks.size(0);
    const float sbs = 1.0f / r; // sub-block-size

    const uint lif_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint sub_id = blockIdx.y * blockDim.y + threadIdx.y;

    if (lif_id >= num_lif || sub_id >= r3)
    {
        return;
    }

    const uint3 bpos = make_uint3(
        (valid_blocks[lif_id] / (ny * nz)) % nx,
        (valid_blocks[lif_id] / nz) % ny,
        valid_blocks[lif_id] % nz);
    const uint3 bsize = make_uint3(indexer.size(0), indexer.size(1), indexer.size(2));
    const uint rx = sub_id / (r * r);
    const uint ry = (sub_id / r) % r;
    const uint rz = sub_id % r;

    // Find all 8 neighbours
    float3 points[8];
    float2 sdf_vals[8];

    sdf_vals[0] = get_sdf(bsize, r, bpos, make_uint3(rx, ry, rz), max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    if (isnan(sdf_vals[0].x))
        return;
    points[0] = make_float3(bpos.x + rx * sbs, bpos.y + ry * sbs, bpos.z + rz * sbs);

    sdf_vals[1] = get_sdf(bsize, r, bpos, make_uint3(rx + 1, ry, rz), max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    if (isnan(sdf_vals[1].x))
        return;
    points[1] = make_float3(bpos.x + (rx + 1) * sbs, bpos.y + ry * sbs, bpos.z + rz * sbs);

    sdf_vals[2] = get_sdf(bsize, r, bpos, make_uint3(rx + 1, ry + 1, rz), max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    if (isnan(sdf_vals[2].x))
        return;
    points[2] = make_float3(bpos.x + (rx + 1) * sbs, bpos.y + (ry + 1) * sbs, bpos.z + rz * sbs);

    sdf_vals[3] = get_sdf(bsize, r, bpos, make_uint3(rx, ry + 1, rz), max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    if (isnan(sdf_vals[3].x))
        return;
    points[3] = make_float3(bpos.x + rx * sbs, bpos.y + (ry + 1) * sbs, bpos.z + rz * sbs);

    sdf_vals[4] = get_sdf(bsize, r, bpos, make_uint3(rx, ry, rz + 1), max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    if (isnan(sdf_vals[4].x))
        return;
    points[4] = make_float3(bpos.x + rx * sbs, bpos.y + ry * sbs, bpos.z + (rz + 1) * sbs);

    sdf_vals[5] = get_sdf(bsize, r, bpos, make_uint3(rx + 1, ry, rz + 1), max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    if (isnan(sdf_vals[5].x))
        return;
    points[5] = make_float3(bpos.x + (rx + 1) * sbs, bpos.y + ry * sbs, bpos.z + (rz + 1) * sbs);

    sdf_vals[6] = get_sdf(bsize, r, bpos, make_uint3(rx + 1, ry + 1, rz + 1), max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    if (isnan(sdf_vals[6].x))
        return;
    points[6] = make_float3(bpos.x + (rx + 1) * sbs, bpos.y + (ry + 1) * sbs, bpos.z + (rz + 1) * sbs);

    sdf_vals[7] = get_sdf(bsize, r, bpos, make_uint3(rx, ry + 1, rz + 1), max_vec_num, indexer, cube_sdf, cube_std, vec_batch_mapping);
    if (isnan(sdf_vals[7].x))
        return;
    points[7] = make_float3(bpos.x + rx * sbs, bpos.y + (ry + 1) * sbs, bpos.z + (rz + 1) * sbs);

    // Find triangle config.
    int cube_type = 0;
    if (sdf_vals[0].x < 0)
        cube_type |= 1;
    if (sdf_vals[1].x < 0)
        cube_type |= 2;
    if (sdf_vals[2].x < 0)
        cube_type |= 4;
    if (sdf_vals[3].x < 0)
        cube_type |= 8;
    if (sdf_vals[4].x < 0)
        cube_type |= 16;
    if (sdf_vals[5].x < 0)
        cube_type |= 32;
    if (sdf_vals[6].x < 0)
        cube_type |= 64;
    if (sdf_vals[7].x < 0)
        cube_type |= 128;

    // Find vertex position on each edge (weighted by sdf value)
    int edge_config = edgeTable[cube_type];
    float4 vert_list[12];

    if (edge_config == 0)
        return;
    if (edge_config & 1)
        vert_list[0] = sdf_interp(points[0], points[1], sdf_vals[0].y, sdf_vals[1].y, sdf_vals[0].x, sdf_vals[1].x);
    if (edge_config & 2)
        vert_list[1] = sdf_interp(points[1], points[2], sdf_vals[1].y, sdf_vals[2].y, sdf_vals[1].x, sdf_vals[2].x);
    if (edge_config & 4)
        vert_list[2] = sdf_interp(points[2], points[3], sdf_vals[2].y, sdf_vals[3].y, sdf_vals[2].x, sdf_vals[3].x);
    if (edge_config & 8)
        vert_list[3] = sdf_interp(points[3], points[0], sdf_vals[3].y, sdf_vals[0].y, sdf_vals[3].x, sdf_vals[0].x);
    if (edge_config & 16)
        vert_list[4] = sdf_interp(points[4], points[5], sdf_vals[4].y, sdf_vals[5].y, sdf_vals[4].x, sdf_vals[5].x);
    if (edge_config & 32)
        vert_list[5] = sdf_interp(points[5], points[6], sdf_vals[5].y, sdf_vals[6].y, sdf_vals[5].x, sdf_vals[6].x);
    if (edge_config & 64)
        vert_list[6] = sdf_interp(points[6], points[7], sdf_vals[6].y, sdf_vals[7].y, sdf_vals[6].x, sdf_vals[7].x);
    if (edge_config & 128)
        vert_list[7] = sdf_interp(points[7], points[4], sdf_vals[7].y, sdf_vals[4].y, sdf_vals[7].x, sdf_vals[4].x);
    if (edge_config & 256)
        vert_list[8] = sdf_interp(points[0], points[4], sdf_vals[0].y, sdf_vals[4].y, sdf_vals[0].x, sdf_vals[4].x);
    if (edge_config & 512)
        vert_list[9] = sdf_interp(points[1], points[5], sdf_vals[1].y, sdf_vals[5].y, sdf_vals[1].x, sdf_vals[5].x);
    if (edge_config & 1024)
        vert_list[10] = sdf_interp(points[2], points[6], sdf_vals[2].y, sdf_vals[6].y, sdf_vals[2].x, sdf_vals[6].x);
    if (edge_config & 2048)
        vert_list[11] = sdf_interp(points[3], points[7], sdf_vals[3].y, sdf_vals[7].y, sdf_vals[3].x, sdf_vals[7].x);

    // Write triangles to array.
    float4 vp[3];
    for (int i = 0; triangleTable[cube_type][i] != -1; i += 3)
    {
#pragma unroll
        for (int vi = 0; vi < 3; ++vi)
        {
            vp[vi] = vert_list[triangleTable[cube_type][i + vi]];
        }
        if (vp[0].w > max_std || vp[1].w > max_std || vp[2].w > max_std)
        {
            continue;
        }
        int triangle_id = atomicAdd(triangles_count, 1);
        if (triangle_id < max_triangles_count)
        {
#pragma unroll
            for (int vi = 0; vi < 3; ++vi)
            {
                triangles[triangle_id][vi][0] = vp[vi].x;
                triangles[triangle_id][vi][1] = vp[vi].y;
                triangles[triangle_id][vi][2] = vp[vi].z;
                triangle_std[triangle_id][vi] = vp[vi].w;
            }
            triangle_flatten_id[triangle_id] = valid_blocks[lif_id];
        }
    }
}

std::vector<torch::Tensor> marching_cubes_sparse_interp_cuda(
    torch::Tensor indexer,           // (nx, ny, nz) -> data_id
    torch::Tensor valid_blocks,      // (K, )
    torch::Tensor vec_batch_mapping, //
    torch::Tensor cube_sdf,          // (M, rx, ry, rz)
    torch::Tensor cube_std,          // (M, rx, ry, rz)
    const std::vector<int> &n_xyz,   // [nx, ny, nz]
    float max_std,                   // Prune all vertices
    int max_n_triangles              // Maximum number of triangle buffer
)
{
    CHECK_INPUT(indexer);
    CHECK_INPUT(valid_blocks);
    CHECK_INPUT(cube_sdf);
    CHECK_INPUT(cube_std);
    CHECK_INPUT(vec_batch_mapping);
    assert(max_n_triangles > 0);

    const int r = cube_sdf.size(1) / 2;
    const int r3 = r * r * r;
    const int num_lif = valid_blocks.size(0);
    const uint max_vec_num = vec_batch_mapping.size(0);

    torch::Tensor triangles = torch::empty({max_n_triangles, 3, 3},
                                           torch::dtype(torch::kFloat32).device(torch::kCUDA));
    torch::Tensor triangle_flatten_id = torch::empty({max_n_triangles}, torch::dtype(torch::kLong).device(torch::kCUDA));
    torch::Tensor triangle_std = torch::empty({max_n_triangles, 3}, torch::dtype(torch::kFloat32).device(torch::kCUDA));

    dim3 dimBlock = dim3(16, 16);
    uint xBlocks = (num_lif + dimBlock.x - 1) / dimBlock.x;
    uint yBlocks = (r3 + dimBlock.y - 1) / dimBlock.y;
    dim3 dimGrid = dim3(xBlocks, yBlocks);

    thrust::device_vector<int> n_output(1, 0);
    meshing_cube<<<dimGrid, dimBlock, 0, at::cuda::getCurrentCUDAStream()>>>(
        indexer.packed_accessor32<int64_t, 3, torch::RestrictPtrTraits>(),
        valid_blocks.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
        vec_batch_mapping.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
        cube_sdf.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
        cube_std.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
        triangles.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        triangle_std.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        triangle_flatten_id.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
        n_output.data().get(), max_n_triangles, max_vec_num,
        n_xyz[0], n_xyz[1], n_xyz[2], max_std);
    hipStreamSynchronize(at::cuda::getCurrentCUDAStream());

    int output_n_triangles = n_output[0];
    if (output_n_triangles < max_n_triangles)
    {
        // Trim output tensor if it is not full.
        triangles = triangles.index({torch::indexing::Slice(torch::indexing::None, output_n_triangles)});
        triangle_flatten_id = triangle_flatten_id.index({torch::indexing::Slice(torch::indexing::None, output_n_triangles)});
        triangle_std = triangle_std.index({torch::indexing::Slice(torch::indexing::None, output_n_triangles)});
    }
    else
    {
        // Otherwise spawn a warning.
        std::cerr << "Warning from marching cube: the max triangle number is too small " << output_n_triangles << " vs " << max_n_triangles << std::endl;
    }

    return {triangles, triangle_flatten_id, triangle_std};
}
